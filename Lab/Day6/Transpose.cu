
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

const int DIM = 32;

// print GB/s
void postprocess(int n, float ms)
{

    printf("%21f\t", n * sizeof(double)*1e-6 / ms );
}
//Read the in matrix using pieces of 32 items.
// naive transpose
__global__ void transposeNaive(double *out,double *in,int BLOCK)
{
  int x = blockIdx.x * DIM + threadIdx.x;
  int y = blockIdx.y * DIM + threadIdx.y;
  int width = gridDim.x * DIM;

  for (int j = 0; j < DIM; j+= BLOCK)
    out[x*width + (y+j)] = in[(y+j)*width + x];   
//each thread executing transpose DIM/BLOCK_i elements from in column into out row. 
}


__global__ void transposeImproved(double *out, double *in,int BLOCK)
{
  __shared__ float tile[DIM][DIM];
    
  int x = blockIdx.x * DIM + threadIdx.x;
  int y = blockIdx.y * DIM + threadIdx.y;
  int width = gridDim.x * DIM;

  for (int j = 0; j < DIM; j += BLOCK)
     tile[threadIdx.y+j][threadIdx.x] = in[(y+j)*width + x];
//use shared memory in order to transpose the matrix and write back to out in row-wise.
  __syncthreads();  //needed in order to ensure that all the writes are performed.

  x = blockIdx.y * DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * DIM + threadIdx.y;

  for (int j = 0; j < DIM; j += BLOCK)
     out[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}
   
void RunTest(int BLOCK,const int nx,const int ny,const int mem_size){
  dim3 dimGrid(nx/DIM, ny/DIM, 1);
  dim3 dimBlock(DIM, BLOCK, 1);
  printf("%d\t",DIM*BLOCK);
  
  double *h_in = (double*)malloc(mem_size);
  double *h_out = (double*)malloc(mem_size);

  double *d_in, *d_out;
  hipMalloc(&d_in, mem_size);
  hipMalloc(&d_out, mem_size);

  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_in[j*nx + i] = i;

  // device
  hipMemcpy(d_in, h_in, mem_size, hipMemcpyHostToDevice);

  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  float ms;

  hipMemset(d_out, 0, mem_size);
  hipEventRecord(startEvent, 0);
  transposeNaive<<<dimGrid, dimBlock>>>(d_out, d_in,BLOCK);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&ms, startEvent, stopEvent);   //milliseconds
  printf("%21f\t",ms);
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
  postprocess(nx * ny, ms);


  hipMemset(d_out, 0, mem_size); //Reset matrix so i don't have to allocate a new one
  hipEventRecord(startEvent, 0);
  transposeImproved<<<dimGrid, dimBlock>>>(d_out, d_in,BLOCK);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&ms, startEvent, stopEvent);
  printf("%21f\t",ms);
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
  postprocess(nx * ny, ms);
  printf("\n");

  // cleanup
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  hipFree(d_out);
  hipFree(d_in);
  free(h_in);
  free(h_out);


}
int main(int argc, char **argv)
{
 printf("# Threads  Naive Transpose(ms)  Naive Bandwidth (GB/s)   Improved Transpose(ms)  Improved Bandwidth (GB/s)\n");
//it's convenient to have num of threads < than elements in a submatrix.
  
  int BLOCK_1 = 2;   //each thread transpose DIM/BLOCK_i elements in the matrix.
  int BLOCK_2 = 16;
  int BLOCK_3 = 32; 				//each block transpose a submatrix of DIMxDIM size 
  const int nx = 8192;
  const int ny = 8192;
  const int mem_size = nx*ny*sizeof(double);

  RunTest(BLOCK_1,nx,ny,mem_size);
  RunTest(BLOCK_2,nx,ny,mem_size);
  RunTest(BLOCK_3,nx,ny,mem_size);
}
