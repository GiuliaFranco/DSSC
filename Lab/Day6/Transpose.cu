
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

const int DIM = 32;

// print GB/s
void postprocess(int n, float ms)
{

    printf("%21f\t", n * sizeof(double)*1e-6 / ms );   //can be multiplied by 2 -> once for reading the matrix and the other
                                                       //for writing.
}
//Read the in matrix using pieces of 32 items.
// naive transpose
__global__ void transposeNaive(double *out,double *in,int BLOCK)
{
  int x = blockIdx.x * DIM + threadIdx.x;
  int y = blockIdx.y * DIM + threadIdx.y;
  
  for (int j = 0; j < DIM; j+= BLOCK)
    out[x*(gridDim.x * DIM) + (y+j)] = in[(y+j)*(gridDim.x * DIM) + x];   
//each thread executing transpose DIM/BLOCK_i elements from in column into out row. 
}


__global__ void transposeImproved(double *out, double *in,int BLOCK)
{
  __shared__ double aux_mat[DIM][DIM];
    
  int x = blockIdx.x * DIM + threadIdx.x;
  int y = blockIdx.y * DIM + threadIdx.y;
  
  for (int j = 0; j < DIM; j += BLOCK)
     aux_mat[threadIdx.y+j][threadIdx.x] = in[(y+j)*(gridDim.x * DIM) + x];
//use shared memory in order to transpose the matrix and write back to out in row-wise.
  __syncthreads();  //needed in order to ensure that all the writes are performed.

  x = blockIdx.y * DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * DIM + threadIdx.y;

  for (int j = 0; j < DIM; j += BLOCK)
     out[(y+j)*(gridDim.x * DIM) + x] = aux_mat[threadIdx.x][threadIdx.y + j];
}
   
void RunTest(int BLOCK,const int nx,const int ny,const int size){
  dim3 dimGrid(nx/DIM, ny/DIM, 1);
  dim3 dimBlock(DIM, BLOCK, 1);
  printf("%d\t",DIM*BLOCK);
  
  double *h_in = (double*)malloc(size);
  double *h_out = (double*)malloc(size);

  double *d_in, *d_out;
  hipMalloc(&d_in, size);
  hipMalloc(&d_out, size);

  // host
  for (int j = 0; j < ny; j++)
    for (int i = 0; i < nx; i++)
      h_in[j*nx + i] = i;

  // device
  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

  // events for timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  float time_m;

//time measures might contain overhead due to kernel lauch     
  hipMemset(d_out, 0, size);
  hipEventRecord(startEvent, 0);
  transposeNaive<<<dimGrid, dimBlock>>>(d_out, d_in,BLOCK);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time_m, startEvent, stopEvent);   //milliseconds
  printf("%21f\t",time_m);
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
  postprocess(nx * ny, time_m);


  hipMemset(d_out, 0, size); //Reset matrix so i don't have to allocate a new one
  hipEventRecord(startEvent, 0);
  transposeImproved<<<dimGrid, dimBlock>>>(d_out, d_in,BLOCK);
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&time_m, startEvent, stopEvent);
  printf("%21f\t",time_m);
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
  postprocess(nx * ny, time_m);
  printf("\n");

  // cleanup
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  hipFree(d_out);
  hipFree(d_in);
  free(h_in);
  free(h_out);


}
int main(int argc, char **argv)
{
 printf("# Threads  Naive Transpose(ms)  Naive Bandwidth (GB/s)   Improved Transpose(ms)  Improved Bandwidth (GB/s)\n");
//it's convenient to have num of threads < than elements in a submatrix.
  
  int BLOCK_1 = 2;   //each thread transpose DIM/BLOCK_i elements in the matrix.
  int BLOCK_2 = 16;
  int BLOCK_3 = 32; 				//each block transpose a submatrix of DIMxDIM size 
  const int nx = 8192;
  const int ny = 8192;
  const int size = nx*ny*sizeof(double);

  RunTest(BLOCK_1,nx,ny,size);
  RunTest(BLOCK_2,nx,ny,size);
  RunTest(BLOCK_3,nx,ny,size);
}
